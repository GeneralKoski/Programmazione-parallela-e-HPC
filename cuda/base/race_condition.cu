
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void increment(int *a_d) {
*a_d +=1; 
//atomicAdd(a_d, 1); 
}

int main(){

int a=0, *a_d;  
hipMalloc((void**)&a_d, sizeof(int));
hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice); 
float elapsedTime; 

hipEvent_t start , stop; 
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord( start , 0 ); 
increment<<<100,100>>>(a_d);
hipEventRecord( stop, 0 ); 
hipEventSynchronize( stop );
hipEventElapsedTime( &elapsedTime , start , stop ); 
hipEventDestroy( start ); 
hipEventDestroy( stop ); 

 printf("GPUTime elapsed: %f seconds\n", elapsedTime/1000.0); 
 hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost); 
 printf("a=%d\n", a);
 hipFree(a_d); 
}

