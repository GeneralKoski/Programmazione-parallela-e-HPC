
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>    //optarg
#include <time.h>

#define BILLION  1000000000L;

void options(int argc, char * argv[]) ;

int n=500000000;    // intervalli
int nblocks=128;    // numero blocchi
int  threadsPerBlock = 1024;
const double  PI = 3.14159265358979323846264338327950288 ;

__global__ void add( float *res1 ) {

       long int tid = threadIdx.x + blockIdx.x * blockDim.x;       

        double h = 1.0 / (double)(gridDim.x * blockDim.x);
        double x = h * ((double)tid - 0.5);
        double pi1 = (1.0 / (1.0 + x*x)); // f1
	res1[tid] = pi1 * 4 * h;

         __syncthreads();
    // for reductions, threadsPerBlock must be a power of 2 // because of the following code
   int i = blockDim.x/2;
   while (i != 0) {
   	 if (threadIdx.x < i)
	 res1[tid] += res1[tid + i];
	 __syncthreads();
	 i /= 2;
   }

}

/************************************************/

int main(int argc, char **argv ) { 

    options(argc, argv);  /* optarg management */

    n=nblocks*threadsPerBlock; 

    float* res1=(float*)malloc(n*sizeof(float));
    float *dev_res1;
    hipMalloc( (void**)&dev_res1, n*sizeof(float) );

    struct timespec t1,t2,t3;
    double wtime, ktime; 
    clock_gettime( CLOCK_REALTIME ,          &t1) ;

    add<<<nblocks,threadsPerBlock>>>( dev_res1 );
    hipDeviceSynchronize();
    clock_gettime( CLOCK_REALTIME ,          &t2) ;

    hipMemcpy( res1, dev_res1, n*sizeof(float), hipMemcpyDeviceToHost ); 

    float total1=0;
    for (long int i=0;i<n;i+=threadsPerBlock)  total1+=res1[i];

    clock_gettime( CLOCK_REALTIME ,          &t3 ) ;

    wtime = (double) ( t3.tv_sec  - t1.tv_sec )
          + (double) ( t3.tv_nsec - t1.tv_nsec )
            / BILLION;


    ktime = (double)  ( t2.tv_sec  - t1.tv_sec )
           + (double) ( t2.tv_nsec - t1.tv_nsec )
            / BILLION;


    fprintf(stderr,"#intervals blocks pi error wtime(s) ktime(s) \n");
    fprintf(stderr,"CUDA, %ld, %d, %d, %.10f, %.10e, %.4f,  %.4f \n", 
          n, nblocks, threadsPerBlock, total1,  fabs(total1 - PI), wtime, ktime);
    hipFree( dev_res1 );

    return 0; 
}


/************************************************/

void options(int argc, char * argv[])
{
  int i;
   while ( (i = getopt(argc, argv, "t:b:h")) != -1) {
        switch (i)
        {
        case 'b':  nblocks         = strtol(optarg, NULL, 10);  break;
        case 't':  threadsPerBlock = strtol(optarg, NULL, 10);  break;
        case 'h':  printf ("\n%s [-b blocks] [-h]",argv[0]); exit(1);
        default:   printf ("\n%s [-b blocks] [-h]",argv[0]);  exit(1);
        }
    }
}

